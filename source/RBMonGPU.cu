#include "types.h"
#include <hip/hip_runtime_api.h>


namespace rbm_on_gpu {

void setDevice(int device) {
    hipSetDevice(device);
}

void start_profiling() {
    hipProfilerStart();
}

void stop_profiling() {
    hipProfilerStop();
}

} // namespace rbm_on_gpu
