#include "hip/hip_runtime.h"
#include "network_functions/PsiAngles.hpp"
#include "quantum_state/PsiDeep.hpp"
#include "spin_ensembles/ExactSummation.hpp"
#include "spin_ensembles/MonteCarloLoop.hpp"
#include "types.h"

namespace rbm_on_gpu {


template<typename Psi_t, typename SpinEnsemble>
pair<Array<complex_t>, Array<complex_t>> psi_angles(const Psi_t& psi, const SpinEnsemble& spin_ensemble) {
    Array<complex_t> result(psi.get_num_units(), psi.gpu);
    Array<complex_t> result_std(psi.get_num_units(), psi.gpu);

    result.clear();
    result_std.clear();

    auto psi_kernel = psi.get_kernel();
    auto result_data = result.data();
    auto result_std_data = result_std.data();

    spin_ensemble.foreach(
        psi,
        [=] __device__ __host__ (
            const unsigned int spin_index,
            const Spins spins,
            const complex_t log_psi,
            typename Psi_t::Angles& angles,
            const double weight
        ) {
            psi_kernel.foreach_angle(spins, angles, [&](const unsigned int j, const complex_t& angle) {
                generic_atomicAdd(&result_data[j], angle);
                generic_atomicAdd(
                    &result_std_data[j],
                    complex_t(
                        angle.real() * angle.real(),
                        angle.imag() * angle.imag()
                    )
                );
            });
        }
    );

    result.update_host();
    result_std.update_host();

    for(auto j = 0u; j < psi.get_num_units(); j++) {
        result[j] /= spin_ensemble.get_num_steps();
        result_std[j] /= spin_ensemble.get_num_steps();

        result_std[j] -= complex_t(
            result[j].real() * result[j].real(),
            result[j].imag() * result[j].imag()
        );
        result_std[j] = complex_t(sqrt(result_std[j].real()), sqrt(result_std[j].imag()));
    }

    return {result, result_std};
}


template pair<Array<complex_t>, Array<complex_t>> psi_angles(const PsiDeep& psi, const ExactSummation& spin_ensemble);
template pair<Array<complex_t>, Array<complex_t>> psi_angles(const PsiDeep& psi, const MonteCarloLoop& spin_ensemble);

} // namespace rbm_on_gpu
